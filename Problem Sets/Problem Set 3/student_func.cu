#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"



__global__ void calcMax1(float* d_max_out,
			   const float* const d_max_in,
			   size_t n){

  if(blockIdx.x*blockDim.x+threadIdx.x >= n) return;

  extern __shared__ float sdata[];
  size_t tid = threadIdx.x;

  // Copy to shared memory
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  sdata[tid] = d_max_in[i]; /**/
  __syncthreads();
  
  for(size_t s=1; s < blockDim.x; s*=2){
    if(tid %(2*s)==0){
      sdata[tid] = max(sdata[tid],sdata[tid+s]);
    }
    __syncthreads();
  }
  
  if(tid==0){
    d_max_out[blockIdx.x] = sdata[0];
  }
}



__global__ void calcMax2(float* d_max_out,
			   const float* const d_max_in,
			   size_t n){

  if(blockIdx.x*blockDim.x+threadIdx.x >= n) return;

  extern __shared__ float sdata[];
  size_t tid = threadIdx.x;

  // Copy to shared memory
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  sdata[tid] = d_max_in[i]; /**/
  __syncthreads();
  
  for(size_t s=1; s < blockDim.x; s*=2){
    size_t index = 2*s*tid;
    if(index<blockDim.x){
      sdata[index] = max(sdata[index],sdata[index+s]);
    }
    __syncthreads();
  }
  
  if(tid==0){
    d_max_out[blockIdx.x] = sdata[0];
  }
}



__global__ void calcMax3(float* d_max_out,
			   const float* const d_max_in,
			   size_t n){

  if(blockIdx.x*blockDim.x+threadIdx.x >= n) return;

  extern __shared__ float sdata[];
  size_t tid = threadIdx.x;

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  sdata[tid] = d_max_in[i]; /**/
  __syncthreads();
  
  for(size_t s=blockDim.x/2; s>0; s>>=1) {
    if(tid < s)
      sdata[tid] = max(sdata[tid],sdata[tid+s]);
    __syncthreads();
  }
  
  if(tid==0){
    d_max_out[blockIdx.x] = sdata[0];
  }
}



__global__ void calcMax4(float* d_max_out,
			   const float* const d_max_in,
			   size_t n){
  extern __shared__ float sdata[];
  size_t tid = threadIdx.x;
  size_t i = blockDim.x * blockIdx.x*2 + tid;
  if(i+blockDim.x >= n) return;
  sdata[tid] = max(d_max_in[i],d_max_in[i+blockDim.x]); /**/
  __syncthreads();
  
  for(size_t s=blockDim.x/2; s>0; s>>=1) {
    if(tid < s)
      sdata[tid] = max(sdata[tid],sdata[tid+s]);
    __syncthreads();
  }
  
  if(tid==0){
    d_max_out[blockIdx.x] = sdata[0];
  }
}


__device__ void warpMax(volatile float* sdata, int tid){
  sdata[tid] = max(sdata[tid],sdata[tid+32]);
  sdata[tid] = max(sdata[tid],sdata[tid+16]);
  sdata[tid] = max(sdata[tid],sdata[tid+ 8]);
  sdata[tid] = max(sdata[tid],sdata[tid+ 4]);
  sdata[tid] = max(sdata[tid],sdata[tid+ 2]);
  sdata[tid] = max(sdata[tid],sdata[tid+ 1]);
}

__global__ void calcMax5(float* d_max_out,
			   const float* const d_max_in,
			   size_t n){
  extern __shared__ float sdata[];
  size_t tid = threadIdx.x;
  size_t i = blockDim.x * blockIdx.x*2 + tid;
  if(i+blockDim.x >= n) return;
  sdata[tid] = max(d_max_in[i],d_max_in[i+blockDim.x]); /**/
  __syncthreads();
  
  for(size_t s=blockDim.x/2; s>32; s>>=1) {
    if(tid < s)
      sdata[tid] = max(sdata[tid],sdata[tid+s]);
    __syncthreads();
  }
  if(tid < 32) warpMax(sdata,tid);
  
  if(tid==0){
    d_max_out[blockIdx.x] = sdata[0];
  }
}



__device__ void warpMin(volatile float* sdata, int tid){
  sdata[tid] = min(sdata[tid],sdata[tid+32]);
  sdata[tid] = min(sdata[tid],sdata[tid+16]);
  sdata[tid] = min(sdata[tid],sdata[tid+ 8]);
  sdata[tid] = min(sdata[tid],sdata[tid+ 4]);
  sdata[tid] = min(sdata[tid],sdata[tid+ 2]);
  sdata[tid] = min(sdata[tid],sdata[tid+ 1]);
}

__global__ void calcMinMax(float* d_min_out,
			   float* d_max_out,
			   const float* const d_min_in,
			   const float* const d_max_in,
			   size_t n)
{
  extern __shared__ float sdata[];
  size_t tid = threadIdx.x;
  size_t i = blockDim.x * blockIdx.x*2 + tid;
  if(i+blockDim.x >= n) return;
  sdata[tid] = max(d_max_in[i],d_max_in[i+blockDim.x]); /**/
  __syncthreads();
  
  for(size_t s=blockDim.x/2; s>32; s>>=1) {
    if(tid < s)
      sdata[tid] = max(sdata[tid],sdata[tid+s]);
    __syncthreads();
  }
  if(tid < 32) warpMax(sdata,tid);
  
  if(tid==0){
    d_max_out[blockIdx.x] = sdata[0];
  }
  sdata[tid] = min(d_min_in[i],d_min_in[i+blockDim.x]); /**/
  __syncthreads();
  
  for(size_t s=blockDim.x/2; s>32; s>>=1) {
    if(tid < s)
      sdata[tid] = min(sdata[tid],sdata[tid+s]);
    __syncthreads();
  }
  if(tid < 32) warpMin(sdata,tid);
  
  if(tid==0){
    d_min_out[blockIdx.x] = sdata[0];
  }
}




void check_d_mem(const float* const d_mem, size_t size, size_t offset){
  float *tmp; // delthis
  tmp = (float*)malloc(size*sizeof(float));
  checkCudaErrors(hipMemcpy(tmp,d_mem,
			     size*sizeof(float),
			     hipMemcpyDeviceToHost));
  //for(size_t i=0 ; i<size; i++) if(tmp[i]!=-4) std::cout << "good i: " << i << std::endl;
  for(size_t i=offset; i<offset+16; i++) std::cout << "tmp[" << i << "]=" << tmp[i] << std::endl;
  std::cout <<std::endl; //**/
  free(tmp);
}



void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  /*
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
  */
  size_t N = 1024;
  int blockSize = N;
  int gridSize = (numCols*numRows)/N;

  std::cout << "numCols: " << numCols;
  std::cout << ", numRows: " << numRows;
  std::cout << ", blockSize: " << blockSize;
  std::cout << ", gridSize: " << gridSize;
  std::cout << ", size: " << numCols*numRows;
  std::cout << std::endl;
  
  




  float *d_max_out,*d_min_out;
  checkCudaErrors(hipMalloc(&d_max_out,numCols*numRows*sizeof(float)));
  checkCudaErrors(hipMalloc(&d_min_out,numCols*numRows*sizeof(float)));
  /*
#define calcMax calcMax5
  calcMax<<<gridSize/2,blockSize,blockSize*sizeof(float)>>>
    (d_max_out,d_logLuminance,numRows*numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  // check_d_mem(d_max_out,numCols*numRows,0);

  calcMax<<<1,gridSize/2,blockSize*sizeof(float)>>>
    (d_max_out,d_max_out,numRows*numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  // check_d_mem(d_max_out,numCols*numRows,0);
  /**/

  calcMinMax<<<gridSize/2,blockSize,blockSize*sizeof(float)>>>
    (d_min_out,d_max_out,d_logLuminance,d_logLuminance,numRows*numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  calcMinMax<<<1,gridSize/2,blockSize*sizeof(float)>>>
    (d_min_out,d_max_out,d_min_out,d_max_out,numRows*numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  float maxLogLum,minLogLum;
  checkCudaErrors(hipMemcpy(&maxLogLum, d_max_out, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&minLogLum, d_min_out, sizeof(float), hipMemcpyDeviceToHost));
  
  std::cout << "---> HI <---" << std::endl;
  std::cout << "MAX: " << maxLogLum << std::endl;
  std::cout << "MIN: " << minLogLum << std::endl;
  std::cout << "DIFF: " << maxLogLum - minLogLum << std::endl;

  // std::cout << std::min_element(h_logLuminance, h_logLuminance+numCols*numRows);
  std::cout << std::endl;
  //// logLumMax = std::max(h_logLuminance[i], logLumMax);

  /*
    //Step 2 
    2) subtract them to find the range
  */
  float logLumRange = max_logLum - min_logLum;

  logLumRange+=1.;
  logLumRange-=1.;
  N = (size_t)d_cdf;
  N= numBins;

  /*
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  //Step 3
  //next we use the now known range to compute
  //a histogram of numBins bins
  /*
  unsigned int *histo = new unsigned int[numBins];

  for (size_t i = 0; i < numBins; ++i) histo[i] = 0;

  for (size_t i = 0; i < numCols * numRows; ++i) {
    unsigned int bin = std::min(static_cast<unsigned int>(numBins - 1),
                           static_cast<unsigned int>((d_logLuminance[i] - min_logLum) / logLumRange * numBins));
    histo[bin]++;
  }

  //Step 4
  //finally we perform and exclusive scan (prefix sum)
  //on the histogram to get the cumulative distribution
  d_cdf[0] = 0;
  for (size_t i = 1; i < numBins; ++i) {
    d_cdf[i] = d_cdf[i - 1] + histo[i - 1];
  }

  delete[] histo;
  */
}
